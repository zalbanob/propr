#include <cfloat>
#include <math.h>
#include <stdio.h>

#include <propr/context.h>
#include <propr/utils.hpp>

#include <propr/kernels/cuda/detail/lrv.cuh>
#include <propr/kernels/cuda/dispatch/lrv.cuh>


using namespace Rcpp;
using namespace propr;

void
dispatch::cuda::lrv_basic(Rcpp::NumericVector& out,
                          Rcpp::NumericMatrix &Y,
                          propr_context context) {
    int N_samples = Y.nrow();
    int N_genes   = Y.ncol();
    int N_pairs = N_genes * (N_genes - 1) / 2;
    CHECK_VECTOR_SIZE(out, N_pairs);

    float* d_Y;
    int stride; d_Y = RcppNumericMatrixToDeviceFloat(Y, stride);
    float* d_variances;
    CUDA_CHECK(hipMalloc(&d_variances, N_pairs * sizeof(float)));

    dim3 blockDim(16, 16);
    dim3 gridDim((N_genes + blockDim.x - 1) / blockDim.x, (N_genes + blockDim.y - 1) / blockDim.y);

    detail::cuda::lrv_basic<<<gridDim, blockDim, 0, context.stream>>>(
        d_Y, stride, d_variances, N_samples, N_genes
    );
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipStreamSynchronize(context.stream));

    copyToNumericVector(d_variances, out, N_pairs);
    CUDA_CHECK(hipFree(d_Y));
    CUDA_CHECK(hipFree(d_variances));
}

void
dispatch::cuda::lrv_weighted(Rcpp::NumericVector& out,
                             Rcpp::NumericMatrix &Y,
                             Rcpp::NumericMatrix &W,
                             propr_context context) {
    int N_samples = Y.nrow();
    int N_genes = Y.ncol();
    int N_pairs = N_genes * (N_genes - 1) / 2;
    CHECK_VECTOR_SIZE(out, N_pairs);

    float* d_Y, * d_W;
    int Y_stride; d_Y = RcppNumericMatrixToDeviceFloat(Y, Y_stride);
    int W_stride; d_W = RcppNumericMatrixToDeviceFloat(W, W_stride);

    float* d_variances;
    CUDA_CHECK(hipMalloc(&d_variances, N_pairs * sizeof(float)));

    dim3 blockDim(16, 16);
    dim3 gridDim((N_genes + blockDim.x - 1) / blockDim.x, (N_genes + blockDim.y - 1) / blockDim.y);

    detail::cuda::lrv_weighted<<<gridDim, blockDim, 0, context.stream>>>(
        d_Y, Y_stride, 
        d_W, W_stride, 
        d_variances, N_samples, N_genes
    );
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipStreamSynchronize(context.stream));

    copyToNumericVector(d_variances, out, N_pairs);

    CUDA_CHECK(hipFree(d_Y));
    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(d_variances));
}

void dispatch::cuda::lrv_alpha(Rcpp::NumericVector& out,
                               Rcpp::NumericMatrix &Y,
                               double a,
                               Rcpp::NumericMatrix& Yfull,
                               propr_context context) {
    int N_samples = Y.nrow();
    int N_genes = Y.ncol();
    int N_pairs = N_genes * (N_genes - 1) / 2;
    CHECK_VECTOR_SIZE(out, N_pairs);

    float* d_Y;
    float* d_Yfull;

    int Y_stride    ; d_Y     = RcppNumericMatrixToDeviceFloat(Y,         Y_stride);
    int Yfull_stride; d_Yfull = RcppNumericMatrixToDeviceFloat(Yfull, Yfull_stride);

    float* d_variances;
    CUDA_CHECK(hipMalloc(&d_variances, N_pairs * sizeof(float)));

    dim3 blockDim(16, 16);
    dim3 gridDim((N_genes + blockDim.x - 1) / blockDim.x, (N_genes + blockDim.y - 1) / blockDim.y);

    detail::cuda::lrv_alpha<<<gridDim, blockDim, 0, context.stream>>>(
        d_Y    , Y_stride,
        d_Yfull, Yfull_stride,
        static_cast<float>(a), d_variances, N_samples, N_genes
    );
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipStreamSynchronize(context.stream));

    copyToNumericVector(d_variances, out, N_pairs);

    CUDA_CHECK(hipFree(d_Y));
    CUDA_CHECK(hipFree(d_Yfull));
    CUDA_CHECK(hipFree(d_variances));
}

void
dispatch::cuda::lrv_alpha_weighted(Rcpp::NumericVector& out,
                                   Rcpp::NumericMatrix &Y,
                                   Rcpp::NumericMatrix &W,
                                   double a,
                                   Rcpp::NumericMatrix& Yfull,
                                   Rcpp::NumericMatrix& Wfull,
                                   propr_context context) {
    int N_samples = Y.nrow();
    int N_genes = Y.ncol();
    int N_pairs = N_genes * (N_genes - 1) / 2;
    CHECK_VECTOR_SIZE(out, N_pairs);
    float* d_Y,* d_W,* d_Yfull,* d_Wfull;
    int Y_stride    ; d_Y     = RcppNumericMatrixToDeviceFloat(Y    , Y_stride    );
    int W_stride    ; d_W     = RcppNumericMatrixToDeviceFloat(W    , W_stride    );
    int Yfull_stride; d_Yfull = RcppNumericMatrixToDeviceFloat(Yfull, Yfull_stride);
    int Wfull_stride; d_Wfull = RcppNumericMatrixToDeviceFloat(Wfull, Wfull_stride);

    float* d_variances;
    CUDA_CHECK(hipMalloc(&d_variances, N_pairs * sizeof(float)));

    dim3 blockDim(16, 16);
    dim3 gridDim((N_genes + blockDim.x - 1) / blockDim.x, (N_genes + blockDim.y - 1) / blockDim.y);

    detail::cuda::lrv_alpha_weighted<<<gridDim, blockDim, 0, context.stream>>>(
        d_Y    , Y_stride, 
        d_Yfull, Yfull_stride, 
        d_W    , W_stride,
        d_Wfull, Wfull_stride,
        static_cast<float>(a), d_variances, N_samples, N_genes
    );
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipStreamSynchronize(context.stream));

    copyToNumericVector(d_variances, out, N_pairs);

    CUDA_CHECK(hipFree(d_Y));
    CUDA_CHECK(hipFree(d_W));
    CUDA_CHECK(hipFree(d_Yfull));
    CUDA_CHECK(hipFree(d_Wfull));
    CUDA_CHECK(hipFree(d_variances));
}