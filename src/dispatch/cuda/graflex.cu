#include "hip/hip_runtime.h"
#include <array>

#include <hip/hip_cooperative_groups.h>

#include <cub/device/device_scan.cuh>

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/transform_reduce.h>
#include <thrust/device_vector.h>

#include <propr/utils.hpp>
#include <propr/kernels/cuda/detail/graflex.cuh>
#include <propr/kernels/cuda/dispatch/graflex.cuh>



using namespace Rcpp;
using namespace propr;

void
dispatch::cuda::getOR(NumericVector& out, 
                      const IntegerMatrix& A, 
                      const IntegerMatrix& G, 
                      propr::propr_context context) {

    using scan_tile_state_t = hipcub::ScanTileState<int4>;
    
    const int n = A.ncol();

    int g_stride; int* d_G; d_G = RcppMatrixToDevice<int>(G, g_stride);
    int a_stride; int* d_A; d_A = RcppMatrixToDevice<int>(A, a_stride);

    const int numPairs      = (n * (n - 1)) / 2;
    const int blockSize     = 512;
    const int numBlocks     = hipcub::DivideAndRoundUp(numPairs, blockSize);
    const int numInitBlocks = hipcub::DivideAndRoundUp(numBlocks, blockSize);

    std::size_t device_partials_size{};
    scan_tile_state_t::AllocationSize(numBlocks, device_partials_size);

    std::size_t aligned_size = ((device_partials_size + 15) / 16) * 16;
    void *d_temp_storage;
    CUDA_CHECK(hipMalloc(&d_temp_storage, aligned_size));
    
    scan_tile_state_t tile_state;
    tile_state.Init(
        numBlocks,
        d_temp_storage,
        device_partials_size
    );

    int4 *d_acc;
    CUDA_CHECK(hipMalloc(&d_acc, sizeof(int4)));
    CUDA_CHECK(hipMemset(d_acc, 0, sizeof(int4)));

    void *kernelArgs[] = {
        (void *)&tile_state,
        (void *)&d_A, 
        (void *)&a_stride,
        (void *)&d_G, 
        (void *)&g_stride,
        (void *)&n,
        (void *)&d_acc
    };
    hipLaunchCooperativeKernel((void *)detail::cuda::compute_odd_ratio, numBlocks, blockSize, kernelArgs, 0, NULL);
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipGetLastError());

    int4 h_acc;
    CUDA_CHECK(hipMemcpy(&h_acc, d_acc, sizeof(int4), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_acc));

    int h_a = h_acc.x;
    int h_b = h_acc.y;
    int h_c = h_acc.z;
    int h_d = h_acc.w;

    double odds_ratio = static_cast<double>(h_a * h_d) / (h_b * h_c);
    double log_odds_ratio = std::log(odds_ratio);
    out[0] = h_a;
    out[1] = h_b;
    out[2] = h_c;
    out[3] = h_d;
    out[4] = odds_ratio;
    out[5] = log_odds_ratio;
    out[6] = R_NaN;
    out[7] = R_NaN;
    
    CUDA_CHECK(hipFree(d_G));
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_temp_storage));
}

void
dispatch::cuda::getORperm(NumericVector& out, const IntegerMatrix& A, const IntegerMatrix& G, const IntegerVector& perm, propr::propr_context context) {
    Rcpp::stop("Not implemented in CUDA for getORperm.");
    int a = 0, b = 0, c = 0, d = 0;
    double odds_ratio = static_cast<double>(a * d) / (b * c);
    double log_odds_ratio = std::log(odds_ratio);
}

void
dispatch::cuda::permuteOR(NumericMatrix& out, const IntegerMatrix& A, const IntegerMatrix& G, int p, propr::propr_context context) {
    Rcpp::stop("Not implemented in CUDA for permuteOR.");
}

void
dispatch::cuda::getFDR(List& out, double actual, const NumericVector& permuted, propr::propr_context context) {
    const int n = permuted.size();
    double* d_permuted;
    CUDA_CHECK(hipMalloc(&d_permuted, n * sizeof(double)));
    CUDA_CHECK(hipMemcpy(
        d_permuted, permuted.begin(), n * sizeof(double),
        hipMemcpyHostToDevice
    ));

    thrust::device_ptr<double> dev_permuted(d_permuted);
    auto policy = thrust::cuda::par.on(context.stream);
    constexpr int2 init{0,0};
    int2 result = thrust::transform_reduce(
        policy,
        dev_permuted,  dev_permuted + n,
        [=] __host__ __device__ (double x) {
            return int2{ x >= actual, x <= actual };
        },
        init,
        [] __host__ __device__ (const int2& a, const int2& b) {
            return int2{ a.x + b.x, a.y + b.y };
        }
    );

    CUDA_CHECK(hipStreamSynchronize(context.stream));
    CUDA_CHECK(hipFree(d_permuted));

    double fdr_over = static_cast<double>(result.x) / n;
    double fdr_under = static_cast<double>(result.y) / n;
    out["over"] = fdr_over;
    out["under"] = fdr_under;
}

void
dispatch::cuda::getG(IntegerMatrix& out, const IntegerVector& Gk, propr::propr_context context) {
    Rcpp::stop("Not implemented in CUDA for getG.");
}

void
dispatch::cuda::graflex(NumericVector& out, const IntegerMatrix& A, const IntegerVector& Gk, int p, propr::propr_context context) {
    Rcpp::stop("Not implemented in CUDA for graflex.");   
}